#include <iostream>
#include "gpu.hpp"

void printCudaVersion()
{
#ifdef OLD_CUDA
    std::cout << "CUDA Compiled version: " << __CUDACC_VER__ << std::endl;
#else
    std::cout << "CUDA Compiled version: " << __CUDACC_VER_MAJOR__ * 10000 + __CUDACC_VER_MAJOR__ * 100 + __CUDACC_VER_BUILD__ << std::endl;
#endif

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}
